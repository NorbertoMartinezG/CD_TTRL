#include "hip/hip_runtime.h"
//////------------------------------ 900 ADVANCED SCENARIOS  ---------------------------------------------------------------------------------  

/*
- Inline PTX
- Driver API
- Pinned Memory (again!)
- Multi-GPU programming
- Thrust

PTX es el lenguaje ensamblador de CUDA
- you can output PTX code from your kernel
	- nvcc -ptx
	- Project setting

- You can also load a PTX kernel in with Driver API
- Embedding PTX into kernel also possible
	- asm("mov.u32 %0, %%laneid;" : "=r"(laneid));
	- Splices the PTX right into your kernel
	- Allows referencing variables


*/



//////------------------------------ 800 EVENTS AND STREAMS  ---------------------------------------------------------------------------------  

/*
- Events
- Event API
- Event example
- Pinned memory
- Streams
- Stream API
- single stream
- multiple streams

802- Events - how to measure performance?

- use profiler (times only kernel duration + other invocations)
- Cuda events (marca de tiempo que se registra en la gpu)
	- event = timestamp
	- Timestamp recorded on the GPU
	- Invoked from the CPU side

803 - Event API
	- hipEvent_t
	- hipEventCreate(&e)
	- hipEventRecord(e, 0)
	- hipEventSynchronize(e)
	- hipEventElapsedTime(&f, start, stop)


// 804 ATOMIC SUM usando events

#include "hip/hip_runtime.h"
#include ""

#include "sm_20_atomic_functions.h"

#include <stdio.h>
#include <iostream>
using namespace std;

__device__ int dSum = 0;

__global__ void sum(int* d)
{
	int tid = threadIdx.x;
	//dSum += d[tid];
	//IMPLEMENTANDO SUMA ATOMICA
	atomicAdd(&dSum, d[tid]);
}


int main()
{
	const int count = 128;
	const int size = sizeof(int) * count;

	int h[count];
	for (int i = 0; i < count; i++)
	{
		h[i] = i + 1;
	}

	int* d;
	hipMalloc(&d, size);
	hipMemcpy(d, h, size, hipMemcpyHostToDevice);
	
	//EVENT--------------------------------------------------------------
	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);

	hipEventRecord(start);
	sum << <1, count >> > (d);
	hipEventRecord(end);
	hipEventSynchronize(end);

	float elapsed;
	hipEventElapsedTime(&elapsed, start, end);
	//EVENT--------------------------------------------------------------
	
	int hSum;
	hipMemcpyFromSymbol(&hSum, HIP_SYMBOL(dSum), sizeof(int));
	//cout << "The sum of numbers from 1 to " << count << " is: " << hSum << endl;
	cout << "the sum of numbers form 1 to " << count << " is " << hSum << " and it took " << elapsed << " milisec" << endl;
	hipFree(d);
	return 0;
}

*/
/*
// 805 Pineed Memory -- memoria anclada

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
using namespace std;

float timeMemory(bool pinned, bool toDevice) 
{
	const int count = 1 << 20;
	const int iterations = 1 << 6;
	const int size = count * sizeof(int);

	hipEvent_t start, end;
	int* h, * d; // memoria apartada en cpu
	float elapsed; //valor transcurrido o tiempo transcurrido 
	hipError_t	status;

	hipEventCreate(&start);
	hipEventCreate(&end);

	hipMalloc(&d, size); // memoria en gpu

	if (pinned)
	{
		hipHostAlloc(&h, size, hipHostMallocDefault);
	}
	else
	{
		h = new int[count];
	}

	hipEventRecord(start);

	for (int i = 0; i < iterations; i++)
	{
		if (toDevice)
		{
			status = hipMemcpy(d, h, size, hipMemcpyHostToDevice);
		}
		else
		{
			status = hipMemcpy(h, d, size, hipMemcpyDeviceToHost);
		}

	}

	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&elapsed, start, end);

	if (pinned)
	{
		hipHostFree(h);
	}
	else
	{
		delete[] h;
	}
	hipFree(d);
	hipEventDestroy(start);
	hipEventDestroy(end);

	return elapsed;

}

int main()
{
	const int count2 = 1 << 20; // a << b = a * (2 ^ b)  << es el operador de bit de desplazamiento a la izquierda.
	const int iterations2 = 1 << 6;
	cout << "count2 = " << count2 << endl;
	cout << "iterations2 = " << iterations2 << endl;

	cout << "From device, paged memory:\t" << timeMemory(false, false) << endl;
	cout << "To device, paged memory:\t" << timeMemory(false, true) << endl;
	cout << "From device, pinned memory:\t" << timeMemory(true, false) << endl;
	cout << "To device, pinned memory:\t" << timeMemory(true, true) << endl;
}
*/
/*
//////------------------------------ 806-7 API STREAMS

hipStream_t
hipStreamCreate(&stream)
kernel <<< blocks, threads, shared, stream >>>
hipMemcpyAsync() -- solo deja copiar en apuntadores (must use pinned memory)
hipStreamSynchronize(stream)
*/
//////------------------------------ 808 EJEMPLO STREAMS

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <cmath>
#include <ctime>
using namespace std;

const int chunkCount = 1 << 20; // conteo constante y global
const int totalCount = chunkCount << 3; //fragmentos

__global__ void kernel(float* a, float* b, float* c) // kernel con entradas a y b y salida c
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x; // identificacion del hilo unidimensional
	
	if (tid < chunkCount)// la identificacion del hilo debe ser menor al fragmento
	{
		c[tid] = erff(a[tid] + b[tid]); //funcion de error
	}
	
}

int main() // RESULTADO DE EJECUCION 11.22 MILISECONS
{
	//hipDeviceProp_t prop;
	//int device;
	//hipGetDevice(&device);
	//hipGetDeviceProperties(&prop, device);
	//if (!prop.deviceOverlap)  // verifica que la gpu sea compatible (en este caso si lo es, revisar: Extensiones-Nsight-windows-system info)
	//{
	//	return 0;
	//}

	hipEvent_t	start, end; // eventos para grabar
	hipEventCreate(&start);
	hipEventCreate(&end);

	hipStream_t stream1, stream2;
	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);

	float* ha, * hb, * hc, * d1a, *d1b, *d1c,* d2a, * d2b, * d2c; // asignar dentro del host y el device
	const int totalSize = totalCount * sizeof(float);
	const int chunkSize = chunkCount * sizeof(float); /*4. La funci�n sizeof()
Para reservar memoria se debe saber exactamente el n�mero de bytes que ocupa cualquier estructura de datos. 
Tal y como se ha comentado con anterioridad, una peculiaridad del lenguaje C es que estos tama�os pueden variar 
de una plataforma a otra. �C�mo sabemos, entonces, cu�ntos bytes reservar para una tabla de, por ejemplo, 10 enteros? 
El propio lenguaje ofrece la soluci�n a este problema mediante la funci�n sizeof().

La funci�n recibe como �nico par�metro o el nombre de una variable, o el nombre de un tipo de datos, y devuelve 
su tama�o en bytes. De esta forma, sizeof(int) devuelve el n�mero de bytes que se utilizan para almacenar un entero. 
La funci�n se puede utilizar tambi�n con tipos de datos estructurados o uniones tal y como se muestra en el siguiente 
programa (que te recomendamos que te descargues, compiles y ejecutes): */

	// allocate memory -- asignar memoria
	hipMalloc(&d1a, chunkSize);
	hipMalloc(&d1b, chunkSize);
	hipMalloc(&d1c, chunkSize);
	hipMalloc(&d2a, chunkSize);
	hipMalloc(&d2b, chunkSize);
	hipMalloc(&d2c, chunkSize);
	hipHostAlloc(&ha, totalSize, hipHostMallocDefault);
	hipHostAlloc(&hb, totalSize, hipHostMallocDefault);
	hipHostAlloc(&hc, totalSize, hipHostMallocDefault);

	// fill a and b
	srand((unsigned)time(0)); // rellenar del lado de host
	for (int i = 0; i < totalCount; i++)
	{
		ha[i] = rand() / RAND_MAX;
		hb[i] = rand() / RAND_MAX;
	}

	hipEventRecord(start, stream1); // grabar evento

	for (int i = 0; i < totalCount; i+= chunkCount*2) //contador que salta entre cada fragmento de datos para enviarlo al kernel
	{
			hipMemcpyAsync(d1a, ha + i, chunkSize, hipMemcpyHostToDevice, stream1); // copia datos cuando el stream este listo
			hipMemcpyAsync(d2a, hb + i+chunkCount, chunkSize, hipMemcpyHostToDevice, stream2);
			hipMemcpyAsync(d1a, hb + i, chunkSize, hipMemcpyHostToDevice, stream1); // copia datos cuando el stream este listo
			hipMemcpyAsync(d2b, hb + i+chunkCount, chunkSize, hipMemcpyHostToDevice, stream2);

			kernel << < chunkCount / 64, 64, 0, stream1 >> > (d1a, d1b, d1c);
			kernel << < chunkCount / 64, 64, 0, stream2 >> > (d2a, d2b, d2c);
			
			hipMemcpyAsync(hc + i, d1c, chunkSize, hipMemcpyDeviceToHost, stream1);
			hipMemcpyAsync(hc + i+chunkCount, d2c, chunkSize, hipMemcpyDeviceToHost, stream2);
	}

	hipStreamSynchronize(stream1);
	hipStreamSynchronize(stream2);
	hipEventRecord(end, stream1);
	hipEventSynchronize(end);

	float elapsed;
	hipEventElapsedTime(&elapsed, start, end);

	cout << "This took " << elapsed << " milisec " << endl;

	hipHostFree(ha);
	hipHostFree(hb);
	hipHostFree(hc);
	hipFree(d1a);
	hipFree(d1b);
	hipFree(d1c);
	hipFree(d2a);
	hipFree(d2b);
	hipFree(d2c);
	hipStreamDestroy(stream1);
	hipStreamDestroy(stream2);



}






//////------------------------------ 700 ATOMIC OPERATIONS  ---------------------------------------------------------------------------------  
/*
SUMMARY
- evita que las operaciones de los hilos sean interrumpidas por otros hilos que deben esperar
-CUDA supports several atomic operations
	-atomicAdd()
	-atomicOr()
	-atomicMin()... etc.

-Atomics incur a heavy performance penalty


x++ is a read-modify-write operation
- Read x into a register
- increment register value
- Write register back into x
- Effectively { temp 0 x; temp = temp + ; x = temp; }

if twoo threads do x++

- Each thread has its own temp (say t1 and t2)
- { t1 = x; t1 = t1+1; x = t1; }
- { t2 = x; t2 = t2+1; x = t2; }
(RACE CONDITION: THE THREAD THAT WRITES TO X FIRST WINS)

703 - atomic functions

Problema: muchos subprocesos acceden a la misma ubicaci�n de memoria
Las operaciones at�micas garantizan que solo un hilo pueda acceder a la ubicaci�n
Alcance de la cuadr�cula!

atomicOp(x,y)
t1 = *x;		//read
t2 = t1 OP y;	//modify
*a = t2;		//write



*/

// 704 ATOMIC SUM
//
//#include "hip/hip_runtime.h"
//#include ""
//
//#include "sm_20_atomic_functions.h"
//
//#include <stdio.h>
//#include <iostream>
//using namespace std;
//
//__device__ int dSum = 0;
//
//__global__ void sum(int* d)
//{
//	int tid = threadIdx.x;
//	//dSum += d[tid];
//	//IMPLEMENTANDO SUMA ATOMICA
//	atomicAdd(&dSum, d[tid]);
//}
//
//
//int main()
//{
//	const int count = 128;
//	const int size = sizeof(int) * count;
//
//	int h[count];
//	for (int i = 0; i < count; i++)
//	{
//		h[i] = i + 1;
//	}
//
//	int* d;
//	hipMalloc(&d, size);
//	hipMemcpy(d, h, size, hipMemcpyHostToDevice);
//	sum << <1, count >> > (d);
//
//	int hSum;
//	hipMemcpyFromSymbol(&hSum, HIP_SYMBOL(dSum), sizeof(int));
//	cout << "The sum of numbers from 1 to " << count << " is: " << hSum << endl;
//
//	hipFree(d);
//	return 0;
//}

//705 Monte carlo Pi.
//
//#include "hip/hip_runtime.h"
//#include ""
//
//#include "hiprand.h"
//
//#include <stdio.h>
//#include <iostream>
//#include <iomanip> // precision numerica
//
//using namespace std;
//
//__device__ int dCount = 0; //la cuenta se inicializa en 0
//
////kernel
//__global__ void countPoints(const float* xs, const float* ys) 
//{
//	int idx = blockIdx.x * blockDim.x + threadIdx.x; // obteniendo indice
//
//	float x = xs[idx] - 0.5f; 
//	float y = ys[idx] - 0.5f;
//	//Evaluacion de la posicion del circulo
//	int n = sqrtf(x * x + y * y) > 0.5f ? 0 : 1; // 0 fuera del circulo, 1 dentro del circulo
//
//	// operacion atomica
//	atomicAdd(&dCount, n);
//
//	//A VECES HAY QUE VERIFICAR QUE hiprand.lib EXISTA (LINKER-INPUT-ADDITIONAL DEPENDENCIES)
//
//}
//
//
//int main()
//{
//	const int count = 512 * 512;
//	const int size = count * sizeof(float);
//	hipError_t cudaStatus;
//	hiprandStatus_t hiprandStatus;
//	hiprandGenerator_t gen; // generador
//
//	//generando matriz de 512 * 512
//	hiprandStatus = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32);
//	hiprandSetPseudoRandomGeneratorSeed(gen, time(0)); // generador de semilla
//
//	//generar 2 matrices
//	float* x;
//	float* y;
//	cudaStatus = hipMalloc(&x, size);
//	cudaStatus = hipMalloc(&y, size);
//
//	//generando datos aleatorios uniformes
//	hiprandStatus = hiprandGenerateUniform(gen, x, count);
//	hiprandStatus = hiprandGenerateUniform(gen, y, count);
//	
//	// contar puntos
//	countPoints << <512, 512 >> > (x, y);
//
//	int hCount;
//	hipMemcpyFromSymbol(&hCount, HIP_SYMBOL(dCount), sizeof(int));
//
//	hipFree(x);
//	hipFree(y);
//
//	cout << setprecision(12) << "pi is aproximately " << (4.0f * (float)hCount) / ((float)count) << endl;
//
//
//}
//


//////------------------------------ 600 THREAD COOPERATION AND SYNCHRONIZATION  ---------------------------------------------------------------------------------  
/*
-Interaccion entre hilos
Los subprocesos pueden tardar diferentes cantidades de tiempo en completar una parte de un c�lculo.
A veces, desea que todos los hilos lleguen a un punto en particular antes de continuar con su trabajo.
Cuda ofrece una funci�n de barrera de hilos __syncthreads ().
Un hilo que llama a __syncthreads () espera a que otros hilos lleguen a esta l�nea.



*/





//////------------------------------501-506 THE MANY TYPES OF MEMORY  ---------------------------------------------------------------------------------  

/*
GRPHICS PROCESSOR ARCHITECTURE

-SM-1
	SP-1
		-Texture cache
		-Constant cache
		-Shared Memory
		-Device Memory
	SP-2
		-Texture cache
		-Constant cache
		-Shared Memory
		-Device Memory
	SP-N
		-Texture cache
		-Constant cache
		-Shared Memory
		-Device Memory
-Device Memory

DEVICE MEMORY

-Grid scope ( available to all threads in all blocks in the grid )
-Aplication lifetime ( una vez que se asigna, existe hasta que se cierra la aplicacion
-Dynamic 
	-hipMalloc() -- Asignar parte de la memoria del dispositivo y luego pasa el puntero
	-Pass pointer to kernel -- pasa el puntero a la memoria del kernel que desea ejecutar
	-hipMemcpy() -- copia desde la memoria del host
	-hipFree() -- Desasigna memoria

-Static
	-Declare global variable as device
		__device__ int sum = 0; -- automaticamente asigna memoria e incluso la inicializa para usarla dentro del kernel
	-Use freely within the kernel
	-Use hipMemcpy[to/from] symbol() to copy to/from host memory
	-No need to explicity deallocate

503- Constant & texture memory
	-Memoria constante -- 64 kb
	-Declare as __constant__
	-hipMemcpy [To/From] Symbol() to copy to/from host memory
	-Es muy util cuando todos los hilos leen la misma ubicacion

504 -Shared Memory
	-compartir solo entre hilos del mismo bloque
	-no se puede compartir entre bloques		
*/

//EJEMPLO SHARED MEMORY CON OPERACION REDUCE

 /*suma paralela
 Sumar todos los elementos en un vector.*/

//#include "hip/hip_runtime.h"
//#include ""
//
//#include <iostream>
//
//using namespace std;
//
//__global__ void sumSingleBlock(int* d) // kernel 
//{
//	extern __shared__ int dcopy[]; //SE AGREGO PARA SHARED COPY
//	int tid = threadIdx.x; //256 sumSingleBlock << <1, count / 2 >> > (d) **count = 512
//	dcopy[tid * 2] = d[tid * 2];		//SE AGREGO PARA SHARED COPY
//	dcopy[tid * 2+1] = d[tid * 2+1];	//SE AGREGO PARA SHARED COPY
//
//	//recuento de hilos  tc - number of participating threads
//	//recuento de subprocesos
//	//blockDim.x = 256 // tc se va dividiendo entre 2 tc >>=1
//	//stepSize aumenta en potencia de 2 stepSize <<= 1 == 1,2,4,8,16,32,64
//	for (int tc = blockDim.x, stepSize = 1; tc > 0; stepSize <<= 1, tc >>=1)   // >>= el valor se desplaza hacia la derecha en uno
//	{
//		//treat must be all
//		if (tid < tc) // 256 < 256   // 
//		{
//			int pa = tid * stepSize * 2; // 256 * 1 * 2 
//			int pb = pa + stepSize;		 // 512 + 1
//			//d[pa] += d[pb];		// d[512] += d[512] + d[513]  
//			dcopy[pa] += dcopy[pb];		// d[512] += d[512] + d[513]  SE AGREGO PARA SHARED COPY
//		}
//
//		// SE AGREGO PARA SHARED COPY
//		if (tid == 0)
//		{
//			d[0] = dcopy[0];
//		}
//	}
//
//}
//
//int main()
//{
//	const int count = 32;
//	const int size = count * sizeof(int);
//
//	int h[count];
//	for (int i = 0; i < count; i++)
//	{
//		h[i] = i + 1;  // rellenar vector origen del 1 al 512
//	}
//
//	// asignar memoria
//	int* d;
//	hipMalloc(&d, size);
//	hipMemcpy(d, h, size, hipMemcpyHostToDevice); // copiar de host a device(gpu)
//
//	// SE AUMENTA ",size" a la instruccion para SHARED MEMORY
//	sumSingleBlock << <1, count / 2, size >> > (d); // 1 = bloque 1 , count/2 = numero de hilos, d = conservar valores de d
//
//	int result;
//	hipMemcpy(&result, d, sizeof(int), hipMemcpyDeviceToHost);
//
//	cout << "Sum is " << result << endl;
//	cout << count << endl;
//	hipFree(d);
//	return 0;
//
//}






//////------------------------------407 Scan  ---------------------------------------------------------------------------------  
// /*suma paralela
// Otra forma de Sumar todos los elementos en un vector.*/
// 
//#include "hip/hip_runtime.h"
//#include ""
//
//#include <iostream>
//
//using namespace std;
//
//__global__ void runningSum(int* d)
//{
//	int threads = blockDim.x;
//	int tid = threadIdx.x;
//
//	//tc - total number of threads allowed
//	for (int tc = threads, step = 1; tc > 0; step *= 2)
//	{
//		// check if thread is allowed to do things
//		if (tid < tc)
//		{
//			d[tid + step] += d[tid];
//		}
//		tc -= step;
//	}
//
//
//
//}
//
//
//int main()
//{
//	const int count = 5;
//	const int size = count * sizeof(int); // esta valor es igual a 64
//
//	int h[count];
//	for (int i = 0; i < count; i++)
//	{
//		h[i] = i + 1;
//	}
//
//	int* d; // contenedor no inicializado
//	hipMalloc(&d, size);
//	hipMemcpy(d, h, size, hipMemcpyHostToDevice);
//
//	runningSum << <1, count - 1 >> > (d);
//
//	hipMemcpy(h, d, size, hipMemcpyDeviceToHost);
//
//	for(int i = 0; i < count; ++i)
//		cout << h[i] << '\t';
//
//	hipFree(d);
//
//	return 0;
//	
//}



////------------------------------406 Reduce ---------------------------------------------------------------------------------  
// suma paralela
// Sumar todos los elementos en un vector.

//#include "hip/hip_runtime.h"
//#include ""
//
//#include <iostream>
//
//using namespace std;
//
//
//__global__ void sumSingleBlock(int* d) // kernel 
//{
//	int tid = threadIdx.x; //256 sumSingleBlock << <1, count / 2 >> > (d) **count = 512
//
//	//recuento de hilos  tc - number of participating threads
//	//recuento de subprocesos
//	//blockDim.x = 256 // tc se va dividiendo entre 2 tc >>=1
//	//stepSize aumenta en potencia de 2 stepSize <<= 1 == 1,2,4,8,16,32,64
//	for (int tc = blockDim.x, stepSize = 1; tc > 0; stepSize <<= 1, tc >>=1)   // >>= el valor se desplaza hacia la derecha en uno
//	{
//		//treat must be all
//		if (tid < tc) // 256 < 256   // 
//		{
//			int pa = tid * stepSize * 2; // 256 * 1 * 2 
//			int pb = pa + stepSize;		 // 512 + 1
//			d[pa] += d[pb];		// d[512] += d[512] + d[513]  
//		}
//	}
//
//
//
//}
//
//
//int main()
//{
//	const int count = 32;
//	const int size = count * sizeof(int);
//
//	int h[count];
//	for (int i = 0; i < count; i++)
//	{
//		h[i] = i + 1;  // rellenar vector origen del 1 al 512
//	}
//
//	// asignar memoria
//	int* d;
//	hipMalloc(&d, size);
//	hipMemcpy(d, h, size, hipMemcpyHostToDevice); // copiar de host a device(gpu)
//
//	sumSingleBlock << <1, count / 2 >> > (d); // 1 = bloque 1 , count/2 = numero de hilos, d = conservar valores de d
//
//	int result;
//	hipMemcpy(&result, d, sizeof(int), hipMemcpyDeviceToHost);
//
//	cout << "Sum is " << result << endl;
//	cout << count << endl;
//	hipFree(d);
//	return 0;
//
//}


////------------------------------404 Gather--recopilar ---------------------------------------------------------------------------------  
// NO FUNCIONA
//BlackScholes

//#include "hip/hip_runtime.h" 
//#include ""
//#include "hiprand.h"
//
//#define _USE_MATH_DEFINES
//#include <iostream>
//#include <ctime>
//#include <cstdio>
//#include <math.h>
//
//using namespace std;
//
//__device__ __host__ __inline__ float N(float x)
//{
//	return 0.5 + 0.5 * erf(x * M_SQRT1_1);
//}
//
//__device__ __host__ void price(float k, float s, float t, float r, float v, float* c, float* p)
//{
//	float srt = v * sqrtf(t);
//	float d1 = (logf(s/k)+(r+0.5*v*v)*t) / srt;
//	float d2 = d1 - srt;
//	float kert = k * expf(-r * t);
//	*c = N(d1) * s - N(d2) * kert;
//	*p = kert - s + *c;
//}
//
//__global__ void price(float* k, float* s, float* t, float* r, float* v, float c, float* p)
//{
//	int idx = threadIdx.x;
//	price(k[idx], s[idx], t[idx], r[idx], v[idx], &c[idx], &p[idx], );
//}
//
//int main()
//{
//	const int count = 512; // numero de elementos a los que debemos poner precio
//	const int size = count * sizeof(float);
//
//	float* args[5];
//	hiprandGenerator_t gen; // Generador
//
//	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32);// inicializador
//	
//	for (int i = 0; i < 5; i++)
//	{
//		hipMalloc(&args[i], size);
//		hiprandGenerateUniform(gen, args[i], count);
//	}
//
//	float* dc, * dp;
//	hipMalloc(&dc, size);
//	hipMalloc(&dp, size);
//
//	price << <1, count >> > (args[0], args[1], args[2], args[3], args[4], dc, dp);
//
//	return 0;
//}


//505 - Resumen

/* - DECLARATION				MEMORY			SCOPE			LIFETIME		SLOWDOWN
	int foo;					register		Thread			kernel			1x
	int foo[10];				local			Thread			kernel			100x
	__shared__ int foo;			Shared			Block			kernel			1x
	__device__ int foo;			global			Grid			Application		100x
	__constant__ int foo;		constant		Grid			Application		1x


*/



////------------------------------401-403 patrones de computacion paralela------------------------------------------------------------------------
// A�ADIR A -- LINKER-INPUT-ADDITIONAL DEPENDECIES -- EN CONFIGURACION LA LIBRERIA CURAND.H
//- Element Addressing
//- Map
//- Gather
//- Scatter
//- Reduce
//- Scan

//Ejemplos
//
//- 1 block, N threads -> htreadldx.x
//- 1 block, MxN threads -> threadldx.y * blockDim.x + threadldx.x
//- N blocks, M threads -> blockldx.x * gridDim.x + threadldx.x

//--  MAP  --
//Aplicar una funcion a cada valor en la entrada

//#include "hip/hip_runtime.h"
//#include ""
//#include "hiprand.h"
//
//#include <iostream>
//#include <ctime>
//#include <cstdio>
//
//
//using namespace std;
//
//
//__global__ void addTen(float* d, int count)
//{
//	int threadsPerBlock = blockDim.x * blockDim.y * blockDim.z; //calculando el indice de un elemento en un espacio de 6 dimensiones(calcular el numero de subprocesos por bloque que existen)
//	int treadPosInBlock = threadIdx.x +  // posicion del hilo (blckDim.x = bloque de dimension) // Tres dimensiones
//		blockDim.x * threadIdx.y +
//		blockDim.x * blockDim.y * threadIdx.z;
//	int blckPosInGrid = blockIdx.x +  // calculo de la posicion del bloque en una cuadricula
//		gridDim.x * blockIdx.y +
//		gridDim.x * gridDim.y * blockIdx.z;
//
//	int tid = blckPosInGrid * threadsPerBlock + treadPosInBlock; // posicion del hilo
//
//	if (tid < count)
//	{
//		d[tid] = d[tid] + 10;
//	}
//
//}	
//
//int main() {
//
//	GENERADOR NUMEROS ALEATORIOS
//	hiprandGenerator_t gen; // genera numeros aleatorios
//	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32); // inicializar generador
//	hiprandSetPseudoRandomGeneratorSeed(gen, time(0));// valor semilla
//	const int cantidad = 123456;//numero de valores a inicializar
//	const int size = cantidad * sizeof(float);
//	float *d; // puntero donde estara almacenado
//	float h[cantidad]; //matriz
//	hipMalloc(&d, size);
//	hiprandGenerateUniform(gen, d, cantidad);
//
//	 dimensiones kernel
//	dim3 block(8, 8, 8); // bloque de 512 
//	dim3 cuadricula(16, 16);
//
//	addTen <<< cuadricula, block >>> (d, cantidad); //inicializamos el kernel
//	
//	hipMemcpy(h, d, size, hipMemcpyDeviceToHost); //copiar valores resultados desde kernel
//
//	hipFree(d); //liberar memoria puntero
//
//	for (int i = 0; i < 100; i++)
//	{
//		cout << h[i] << endl;
//	}
//
//	return 0;
//}


////------------------------------308 Devide Introspection----------------------------------------------------

//#include "hip/hip_runtime.h"
//#include ""
//#include <stdio.h>
//#include <iostream>
//using namespace std;
//
//int main() {
//	int count;
//	hipGetDeviceCount(&count);  //numero de dispositivos o gpu's
//
//	hipDeviceProp_t prop;	//informacion sobre el dispositivo
//
//	for (int i = 0; i < count; i++)
//	{
//		hipGetDeviceProperties(&prop, i);
//		cout << "Device " << i << ": " << prop.name << endl; // nombre del dispositivo
//		cout << "Compute capability: " << prop.major << "." << prop.minor << endl; // capacidad de calculo
//
//		cout << "Maximum grid dimensions: (:" <<
//			prop.maxGridSize[0] << " x " <<
//			prop.maxGridSize[1] << " x " <<
//			prop.maxGridSize[2] << ") " << endl; // dimensiones maximas de cuadricula y bloque
//
//		cout << "Maximum block dimensions: (:" <<
//			prop.maxThreadsDim[0] << " x " <<
//			prop.maxThreadsDim[1] << " x " <<
//			prop.maxThreadsDim[2] << ") " << endl; // dimensiones maximas de cuadricula y bloque
//
//
//
//	}
//
//	return 0;
//}


////------------------------------306-7 Error handling----------------------------------------------------
/*
Para definir la ejecucion
<<<a,b>>>
a = blocks
b = threads
Realmente son 3 dimensiones (a x b x c)

dim3
conversion automatica de <<<a,b>>> = (a,1,1) por (b,1,1)

blockldx	=		donde estamos en la cuadr�cula
gridDim		=		tama�o de la cuadricula
threadldmx	=		posici�n del hilo actual en el bloque de hilo
blockDim	=		Tama�o del bloque de hilo

max_threads_per_block =	 512
max_threads_per_multiprocessor = 1024

VERIFICAR ERRORES EN GPU. (regularmente no los marca a menos que se supervicen los status)
hipSuccess
hipGetErrorString()
cuRAND tiene hiprandStatus_t


*/


//------------------------------303-4 EXECUTION MODEL----------------------------------------------------
/*

LOCATION QUALIFIERS
___global___ === Define el Kernel, corre en la GPU, se llama desde CPU, recibe argumentos <<<dim3>>>
___device___ ===				 , corre en la GPU, se llama desde GPU, se pueden definir variables dentro de GPU
___host___   ===				 , corre en la CPU , se llama desde CPU

 -SE PUEDEN MEZCLAR

EXECUTION MODEL
 sumArrayGpu << <1, count >> > (da, db, dc); // el error se�alado aqui no es un error realmente es fallo de analisis del VS
 //count hace referencia a los hilos EL 1 SE REFIERE A UN BLOQUE Y EL COUNT =5 SE REFIERE A 5 HILOS


*/


//
//#include "hip/hip_runtime.h"
//#include ""
//#include <stdio.h>
//#include <iostream>
//using namespace std;
//
//
//void sumArray(int* a1, int* b1, int* c1, int count1) {
//	for (int i = 0; i < count1; i++)
//	{
//		c1[i] = a1[i] + b1[i];
//	}
//}
//
////modificanco para usar hilos en gpu global se invoca en el CPU pero se ejecuta en GPU
//__global__ void sumArrayGpu(int* a, int* b, int* c) { //GLOBAL -- FUNCION HACIA GPU	
//	int i = threadIdx.x; // indice de hilo
//	c[i] = a[i] + b[i];
//}
//
//void main()
//{
//	const int count1 = 5;
//	int a1[] = { 1,2,3,4,5 };
//	int b1[] = { 10,20,30,40,50 };
//	int c1[count1];
//
//	sumArray(a1, b1, c1, count1);
//
//	//Imprimir
//	for (int i = 0; i < count1; i++)
//	{
//		cout << "posicion: " << i << " corresponde a: " << c1[i] << endl;
//	}
//
//	//----- LO MISMO PERO CON GPU ---
//
//	const int count = 5;
//	const int size = count * sizeof(int); //numero de elementos multiplicado por su tama�o para asignar memoria en GPU
//	//int a[] = { 1,2,3,4,5 };
//	int ha[] = { 1,2,3,4,5 }; // el cambio de nombre es para especificar que esta en el HOST
//	//int b[] = { 10,20,30,40,50 };
//	int hb[] = { 10,20,30,40,50 };
//	//int c[count];
//	int hc[count];
//
//	int* da, * dc, * db;//asignando memoria CPU
//	hipMalloc(&da, size);//asignando memoria en GPU
//	hipMalloc(&db, size);//asignando memoria en GPU
//	hipMalloc(&dc, size);//asignando memoria en GPU
//
//	//copiando datos a GPU
//	//hipMemcpy(da, ha, size, hipMemcpyKind::hipMemcpyHostToDevice); // instruccion completa
//	hipMemcpy(da, ha, size, hipMemcpyHostToDevice); // instruccion corta
//	hipMemcpy(db, hb, size, hipMemcpyHostToDevice); // instruccion corta
//
//	//el 1 se refiere a un bloque
//	//count hace referencia a los hilos EL 1 SE REFIERE A UN BLOQUE Y EL COUNT =5 SE REFIERE A 5 HILOS
//	sumArrayGpu << <1, count >> > (da, db, dc); // el error se�alado aqui no es un error realmente es fallo de analisis del VS
//
//	//Recuperando datos desde GPU
//	hipMemcpy(hc, dc, size, hipMemcpyDeviceToHost); //
//
//	//Imprimir
//	for (int i = 0; i < count1; i++)
//	{
//		cout << "posicion: " << i << " corresponde a: " << hc[i] << endl;
//	}
//
//}


////------------------------------303 HELLO CUDA----------------------------------------------------
//
//#include "hip/hip_runtime.h"
//#include ""
//#include <stdio.h>
//#include <iostream>
//using namespace std;
//
//
//void sumArray(int* a1, int* b1, int* c1, int count1) {
//	for (int i = 0; i < count1; i++)
//	{
//		c1[i] = a1[i] + b1[i];
//	}
//}
//
////modificanco para usar hilos en gpu global se invoca en el CPU pero se ejecuta en GPU
//__global__ void sumArrayGpu(int* a, int* b, int* c) { //GLOBAL -- FUNCION HACIA GPU	
//	int i = threadIdx.x; // indice de hilo
//	c[i] = a[i] + b[i];
//}
//
//void main()
//{
//	const int count1 = 5;
//	int a1[] = { 1,2,3,4,5 };
//	int b1[] = { 10,20,30,40,50 };
//	int c1[count1];
//
//	sumArray(a1, b1, c1, count1);
//
//	//Imprimir
//	for (int i = 0; i < count1; i++)
//	{
//		cout << "posicion: " << i << " corresponde a: " << c1[i] << endl;
//	}
//
//	//----- LO MISMO PERO CON GPU ---
//
//	const int count = 5;
//	const int size = count * sizeof(int); //numero de elementos multiplicado por su tama�o para asignar memoria en GPU
//	//int a[] = { 1,2,3,4,5 };
//	int ha[] = { 1,2,3,4,5 }; // el cambio de nombre es para especificar que esta en el HOST
//	//int b[] = { 10,20,30,40,50 };
//	int hb[] = { 10,20,30,40,50 };
//	//int c[count];
//	int hc[count];
//
//	int* da, * dc, * db;//asignando memoria CPU
//	hipMalloc(&da, size);//asignando memoria en GPU
//	hipMalloc(&db, size);//asignando memoria en GPU
//	hipMalloc(&dc, size);//asignando memoria en GPU
//
//	//copiando datos a GPU
//	//hipMemcpy(da, ha, size, hipMemcpyKind::hipMemcpyHostToDevice); // instruccion completa
//	hipMemcpy(da, ha, size, hipMemcpyHostToDevice); // instruccion corta
//	hipMemcpy(db, hb, size, hipMemcpyHostToDevice); // instruccion corta
//
//	//el 1 se refiere a un bloque
//	//count hace referencia a los hilos
//	sumArrayGpu << <1, count >> > (da, db, dc); // el error se�alado aqui no es un error realmente es fallo de analisis del VS
//
//	//Recuperando datos desde GPU
//	hipMemcpy(hc, dc, size, hipMemcpyDeviceToHost); //
//
//	//Imprimir
//	for (int i = 0; i < count1; i++)
//	{
//		cout << "posicion: " << i << " corresponde a: " << hc[i] << endl;
//	}
//
//}


//
////------------------------------PLANTILLA POR DEFECTO---------------------------------------------
//#include "hip/hip_runtime.h"
//#include ""
//
//#include <stdio.h>
//
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
//
//__global__ void addKernel(int *c, const int *a, const int *b)
//{
//    int i = threadIdx.x;
//    c[i] = a[i] + b[i];
//}
//
//int main()
//{
//    const int arraySize = 5;
//    const int a[arraySize] = { 1, 2, 3, 4, 5 };
//    const int b[arraySize] = { 1000, 20, 30, 40, 50 };
//    int c[arraySize] = { 0 };
//
//    // Add vectors in parallel.
//    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addWithCuda failed!");
//        return 1;
//    }
//
//    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",c[0], c[1], c[2], c[3], c[4]);
//
//    // hipDeviceReset must be called before exiting in order for profiling and
//    // tracing tools such as Nsight and Visual Profiler to show complete traces.
//    cudaStatus = hipDeviceReset();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceReset failed!");
//        return 1;
//    }
//
//    return 0;
//}
//
//// Helper function for using CUDA to add vectors in parallel.
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//    int *dev_a = 0;
//    int *dev_b = 0;
//    int *dev_c = 0;
//    hipError_t cudaStatus;
//
//    // Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = hipSetDevice(0);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    // Launch a kernel on the GPU with one thread for each element.
//    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = hipGetLastError();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
//    
//    return cudaStatus;
//}
